
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {

  int c;
  hipGetDeviceCount(&c);
  printf("Total device %d\n",c);

  int i;
  hipDeviceProp_t deviceProp;
  for(i=0; i<c; i++){
    hipGetDeviceProperties(&deviceProp, i);
    printf("Device %d has compute capability %d.%d.\n",
      i, deviceProp.major, deviceProp.minor);
  }
}
